#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#define KERNEL_SIZE 3
#define TILE_SIZE 30
#define BLOCK_SIZE (TILE_SIZE)+(KERNEL_SIZE)-1

__constant__ float Mc[KERNEL_SIZE][KERNEL_SIZE];

__global__ void 3DConv(float* N,float* P, int height, int width){


}
int main(int argc, const char** argv){
    
    int row_i = atoi(argv[1]);
    int col_i = atoi(argv[2]);
    dim3 dimGrid(ceil(row_i/(TILE_SIZE*1.0)), ceil(col_i/(TILE_SIZE*1.0)),1);
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE,1);



    float *N = (float*)malloc(sizeof(float)*row_i*col_i);
    float M[KERNEL_SIZE][KERNEL_SIZE];
	float* M_ver = (float*)malloc(sizeof(float)*KERNEL_SIZE*KERNEL_SIZE);
    float *result = (float*)malloc(sizeof(float)*row_i*col_i);

	hipMemcpyToSymbol(HIP_SYMBOL(Mc),M,sizeof(float)*KERNEL_SIZE*KERNEL_SIZE);
	hipMemcpy(N_D,N,row_i*col_i*sizeof(float),hipMemcpyHostToDevice);
    

    Conv<<<dimGrid,dimBlock>>>(N_D,result_D,row_i,col_i);
	hipMemcpy(result,result_D,row_i*col_i*sizeof(float),hipMemcpyDeviceToHost);
	verification(N, M_ver, result, row_i, col_i);

	hipFree(N_D);
	hipFree(result_D);
	hipFree(Mc);
	free(N);
	free(M_ver);
	free(result);

	return 0;
}
