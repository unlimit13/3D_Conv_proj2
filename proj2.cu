#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#define KERNEL_SIZE 3
#define TILE_SIZE 30
#define BLOCK_SIZE (TILE_SIZE)+(KERNEL_SIZE)-1

__constant__ float Mc[KERNEL_SIZE][KERNEL_SIZE];

void single_3DConv(){

}
void multi_3DConv(){
	
}
__global__ void 3DConv(float* N,float* P, int height, int width){


}
int main(int argc, const char** argv){
  int state, state2, state3;
    if(argc == 4){
        FILE *input_file = fopen(argv[1],"rt");
        FILE *kernel_file = fopen(argv[2],"rt");
        FILE *output_file = fopen(argv[3],"rt");
        if (input_file == NULL || kernel_file == NULL || output_file == NULL){
            printf("스트림 생성시 오류발생");
            return 1;
       }
       char a;
       while(1){
            if (feof(kernel_file) != 0){
                printf("복사가 완료되었습니다.\n");
                break;
            }


            a = fgetc(kernel_file);
            printf("%c ",a);
       }
       state = fclose(input_file);
        state2 = fclose(kernel_file);
        state3 = fclose(output_file);
    }else{
        printf("parameter 부족\n");
    }
    
    if (state != 0 || state2 != 0 || state3 != 0){
        printf("스트림 제거시 오류발생");
        return 1;
    }
    return 0;

}
