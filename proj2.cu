
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define TILE_SIZE 4
#define KERNEL_SIZE 5
#define BLOCK_SIZE (TILE_SIZE)-1
void verification(float* GPU, float* output, int height, int col, int row){
    for(int i=0;i<height;i++){
        for(int j=0;j<col;j++){
            for(int k=0;k<row;k++){
                printf("GPU : %f vs output : %f \n",GPU[i*(row*col)+j*row+k],output[i*(row*col)+j*row+k]);
                if(abs(GPU[i*(row*col)+j*row+k]-output[i*(row*col)+j*row+k]) < 0.001f){
                    printf("---\n");
                }
                else{
                    printf("NON EQUAL\n");
                }
            }
        }
    }

}

__constant__ float Kernel_const[KERNEL_SIZE*KERNEL_SIZE*KERNEL_SIZE];
__global__ void __3DConv(float* input_D,float* output_D,int height,int col,int row,int size){
	int tx=threadIdx.x;
    int ty=threadIdx.y;
    int tz=threadIdx.z;

    int row_o = blockIdx.x*TILE_SIZE+tx;
    int col_o = blockIdx.y*TILE_SIZE+ty;
    int hei_o = blockIdx.z*TILE_SIZE+tz;

    int row_i = row_o-((size-1)/2);
	int col_i = col_o-((size-1)/2);
    int hei_i = hei_o-((size-1)/2);


    float output=0.0f;
    __shared__ float input_slice[TILE_SIZE+KERNEL_SIZE-1][TILE_SIZE+KERNEL_SIZE-1][TILE_SIZE+KERNEL_SIZE-1];
	if((row_i>=0)&&(row_i<row)&&(col_i>=0)&&(col_i<col)&&(hei_i>=0)&&(hei_i<height)){
		input_slice[tz][ty][tx]=input_D[hei_i*(row*col)+col_i*row+row_i];
                      

	}
	else{
		input_slice[tz][ty][tx] = 0.0f;
	}
    __syncthreads();
    if(tz < TILE_SIZE && ty < TILE_SIZE && tx < TILE_SIZE){
		for(int i = 0; i < size; i++){
			for(int j = 0; j < size; j++){
				for(int k = 0; k < size; k++){
                    output += Kernel_const[i*(KERNEL_SIZE*KERNEL_SIZE)+j*(KERNEL_SIZE)+k] * input_slice[i+tz][j+ty][k+tx];
                }
			}
		}

        //__syncthreads();
		// some threads do not write output
		if (hei_o < height && col_o < col && row_o < row){
			output_D[hei_o*(row*col)+col_o*row+row_o] = output;
		}
	}
}
int main(int argc, const char** argv){
    int state, state2, state3,size,row,col,height;
    float *input, *kernel, *output; 
    if(argc == 4){
        FILE *input_file = fopen(argv[1],"rt");
        FILE *kernel_file = fopen(argv[2],"rt");
        FILE *output_file = fopen(argv[3],"rt");
        if (input_file == NULL || kernel_file == NULL || output_file == NULL){
            printf("스트림 생성시 오류발생");
            return 1;
       }
       char buffer[20],row_temp[20],col_temp[20],height_temp[20];
       fscanf(input_file,"%s",height_temp);
       fscanf(input_file,"%s",col_temp);
       fscanf(input_file,"%s",row_temp);
       height = atoi(height_temp);
       col = atoi(col_temp);
       row = atoi(row_temp);
       
       input = (float*)malloc(sizeof(float) * height*col*row);
       float num;
       for(int i=0;i<height;i++){
           for(int j=0;j<col;j++){
               for(int k=0;k<row;k++){
                    if (feof(input_file) != 0){
                        break;
                    }
                    fscanf(input_file,"%s",buffer);
                    num = atof(buffer);
                    input[i*(row*col)+j*row+k] = num;
                    
               }
           }
       }
       //input

       //kernel
       fscanf(kernel_file,"%s",height_temp);
       size = atoi(height_temp);
       kernel = (float*)malloc(sizeof(float)*KERNEL_SIZE*KERNEL_SIZE*KERNEL_SIZE);
  
        for(int i=0;i<size;i++){
            for(int j=0;j<size;j++){
                for(int k=0;k<size;k++){
                    if (feof(kernel_file) != 0){
                        break;
                    }
                    fscanf(kernel_file,"%s",buffer);
                    num = atof(buffer);
                    kernel[i*(KERNEL_SIZE*KERNEL_SIZE)+j*(KERNEL_SIZE)+k] = num;
                }
            }
        }
        //kernel

        //output
        fscanf(output_file,"%s",height_temp);
        fscanf(output_file,"%s",col_temp);
        fscanf(output_file,"%s",row_temp);
        height = atoi(height_temp);
        col = atoi(col_temp);
        row = atoi(row_temp);

        output = (float*)malloc(sizeof(float) * height*col*row);


       for(int i=0;i<height;i++){
           for(int j=0;j<col;j++){
               for(int k=0;k<row;k++){
                    if (feof(output_file) != 0){
                        break;
                    }
                    fscanf(output_file,"%s",buffer);
                    num = atof(buffer);
                    output[i*(row*col)+j*row+k] = num;
               }
           }
       }
    //output
       
       state = fclose(input_file);
       state2 = fclose(kernel_file);
       state3 = fclose(output_file);
    }
	else{
        printf("parameter 부족\n");
    }
    
    if (state != 0 || state2 != 0 || state3 != 0){
        printf("스트림 제거시 오류발생");
        return 1;
    }

	dim3 dimGrid(ceil(row/(TILE_SIZE*1.0)),ceil(col/(TILE_SIZE*1.0)),ceil(height/(TILE_SIZE*1.0)));
	dim3 dimBlock(BLOCK_SIZE+size,BLOCK_SIZE+size,BLOCK_SIZE+size);


	hipEvent_t start, end;
	float time_ms_GPU=0;
	hipEventCreate(&start);
	hipEventCreate(&end);


    float *input_D,*output_D,*output_result;
    output_result = (float*)malloc(sizeof(float)*height*col*row);
    memset(output_result, 0, height*col*row*sizeof(float));

    hipMalloc((void**)&input_D,sizeof(float)*height*col*row);
    hipMemcpy(input_D,input,sizeof(float)*height*col*row,hipMemcpyHostToDevice);
    hipMalloc((void**)&output_D,sizeof(float)*height*col*row);
    hipMemcpy(output_D,output_result,sizeof(float)*height*col*row,hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(Kernel_const),kernel,sizeof(float)*KERNEL_SIZE*KERNEL_SIZE*KERNEL_SIZE);


	hipEventRecord(start,0);
	__3DConv<<<dimGrid,dimBlock>>>(input_D,output_D,height, col, row, size);
    hipDeviceSynchronize();
	hipMemcpy(output_result,output_D,row*col*height*sizeof(float),hipMemcpyDeviceToHost);
    hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time_ms_GPU,start,end);
    printf("\nExecution time for kernel: %.2f ms\n",time_ms_GPU);

    verification(output_result,output,height,col,row);


    return 0;

}
