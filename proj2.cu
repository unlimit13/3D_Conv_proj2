
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define KERNEL_SIZE 3

__constant__ float Mc[KERNEL_SIZE][KERNEL_SIZE];

void single_3DConv(){

}
void multi_3DConv(){

}
__global__ void _3DConv(){
	__syncthreads();

}
int main(int argc, const char** argv){
    int state, state2, state3;
    float ***input, ***kernel, ***output; 
    if(argc == 4){
        FILE *input_file = fopen(argv[1],"rt");
        FILE *kernel_file = fopen(argv[2],"rt");
        FILE *output_file = fopen(argv[3],"rt");
        if (input_file == NULL || kernel_file == NULL || output_file == NULL){
            printf("스트림 생성시 오류발생");
            return 1;
       }
       char buffer[20],row_temp[20],col_temp[20],height_temp[20];
       int row,col,height;
       fscanf(input_file,"%s",height_temp);
       fscanf(input_file,"%s",col_temp);
       fscanf(input_file,"%s",row_temp);
       height = atoi(height_temp);
       row = atoi(row_temp);
       col = atoi(col_temp);
       
       input = (float***)malloc(sizeof(float**) * height);
       for(int i=0; i<height; i++){
           input[i] = (float**)malloc(sizeof(float*) * col);
           for(int j=0; j<col; j++){
                input[i][j] = (float*)malloc(sizeof(float) * row);
           }
       }

       float num;
       for(int i=0;i<height;i++){
           for(int j=0;j<col;j++){
               for(int k=0;k<row;k++){
                    if (feof(input_file) != 0){
                        break;
                    }
                    fscanf(input_file,"%s",buffer);
                    num = atof(buffer);
                    input[i][j][k] = num;
               }
           }
       }
       //input

       //kernel
       fscanf(kernel_file,"%s",height_temp);
       height = atoi(height_temp);
       kernel = (float***)malloc(sizeof(float**) * height);
       for(int i=0; i<height; i++){
            kernel[i] = (float**)malloc(sizeof(float*) * height);
           for(int j=0; j<height; j++){
                kernel[i][j] = (float*)malloc(sizeof(float) * height);
           }
       }
        for(int i=0;i<height;i++){
            for(int j=0;j<height;j++){
                for(int k=0;k<height;k++){
                    if (feof(kernel_file) != 0){
                        break;
                    }
                    fscanf(kernel_file,"%s",buffer);
                    num = atof(buffer);
                    kernel[i][j][k] = num;
                }
            }
        }
        //kernel

        //output
        fscanf(output_file,"%s",height_temp);
        fscanf(output_file,"%s",col_temp);
        fscanf(output_file,"%s",row_temp);
        height = atoi(height_temp);
        row = atoi(row_temp);
        col = atoi(col_temp);

        output = (float***)malloc(sizeof(float**) * height);
       for(int i=0; i<height; i++){
        output[i] = (float**)malloc(sizeof(float*) * col);
           for(int j=0; j<col; j++){
                output[i][j] = (float*)malloc(sizeof(float) * row);
           }
       }

       for(int i=0;i<height;i++){
           for(int j=0;j<col;j++){
               for(int k=0;k<row;k++){
                    if (feof(output_file) != 0){
                        break;
                    }
                    fscanf(output_file,"%s",buffer);
                    num = atof(buffer);
                    output[i][j][k] = num;
               }
           }
       }
       printf("%f \n",output[0][0][0]);
       printf("%f \n",output[0][0][1]);
       printf("%f \n",output[0][0][2]);
       
       state = fclose(input_file);
       state2 = fclose(kernel_file);
       state3 = fclose(output_file);
    }
	else{
        printf("parameter 부족\n");
    }
    
    if (state != 0 || state2 != 0 || state3 != 0){
        printf("스트림 제거시 오류발생");
        return 1;
    }

	dim3 dimGrid(1,1,1);
	dim3 dimBlock(1,1,1);

	hipEvent_t start, end;
	float time_ms_single=0, time_ms_multi=0, time_ms_GPU=0;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start,0);
	single_3DConv();
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time_ms_single,start,end);

	hipEventRecord(start,0);
	multi_3DConv();
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time_ms_multi,start,end);

	hipEventRecord(start,0);
	_3DConv<<<dimGrid,dimBlock>>>();
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time_ms_GPU,start,end);

    return 0;

}
