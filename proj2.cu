
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define KERNEL_SIZE 3

__constant__ float Mc[KERNEL_SIZE][KERNEL_SIZE];

void single_3DConv(){

}
void multi_3DConv(){

}
__global__ void _3DConv(){
	__syncthreads();

}
int main(int argc, const char** argv){
  int state, state2, state3;
    if(argc == 4){
        FILE *input_file = fopen(argv[1],"rt");
        FILE *kernel_file = fopen(argv[2],"rt");
        FILE *output_file = fopen(argv[3],"rt");
        if (input_file == NULL || kernel_file == NULL || output_file == NULL){
            printf("스트림 생성시 오류발생");
            return 1;
       }
       char a;
       while(1){
            if (feof(kernel_file) != 0){
                printf("복사가 완료되었습니다.\n");
                break;
            }


            a = fgetc(kernel_file);
            printf("%c ",a);
       }
       state = fclose(input_file);
       state2 = fclose(kernel_file);
       state3 = fclose(output_file);
    }
	else{
        printf("parameter 부족\n");
    }
    
    if (state != 0 || state2 != 0 || state3 != 0){
        printf("스트림 제거시 오류발생");
        return 1;
    }

	dim3 dimGrid(1,1,1);
	dim3 dimBlock(1,1,1);

	hipEvent_t start, end;
	float time_ms_single=0, time_ms_multi=0, time_ms_GPU=0;
	hipEventCreate(&start);
	hipEventCreate(&end);

	hipEventRecord(start,0);
	single_3DConv();
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time_ms_single,start,end);

	hipEventRecord(start,0);
	multi_3DConv();
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time_ms_multi,start,end);

	hipEventRecord(start,0);
	_3DConv<<<dimGrid,dimBlock>>>();
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time_ms_GPU,start,end);

    return 0;

}
